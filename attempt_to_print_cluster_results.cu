#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdio>
#include <algorithm>
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include "../common/book.h"

using namespace std;

/*
	Column #: Data
	4: total_cases
	5: new_cases
	7: total_deaths
	8: new_deaths
	10: total_cases_per_million
	11: new_cases_per_million
	13: total_deaths_per_million
	14: new_deaths_per_million
	17: icu_patients
	18: icu_patients_per_million
	19: hosp_patients
	20: hosp_patients_per_million
	25: new_tests
	26: total_tests
	31: positive_rate
	32: tests_per_case
	34: total_vaccinations
	35: people_vaccinated
	36: people_fully_vaccinated
	37: total_boosters
	38: new_vaccinations
	48: median_age
*/

const string FILE_NAME = "Data/owid-covid-data-2021-11-03.csv"; // File location
const int COLUMN_NUMBER_1 = 4; // Column #
const int COLUMN_NUMBER_2 = 7; // Column #
const int COLUMN_NUMBER_3 = 31; // Column #
const int COLUMN_NUMBER_4 = 35; // Column #
const int NUM_RECORDS = 121955; // Number of Records to read
const int NUM_CLUSTERS = 5;
const int NUM_PAIRS = 6; //Number of possible Column pairings
const int NUM_COLUMNS = 4; //Number of columns we are using

void build_input(float input_1[], float input_2[], float input_3[], float input_4[], int locations[]){
	ifstream file;
	int lines = -1, location = 0, column_count = 0;
	string line, current_location, value;
	file.open(FILE_NAME);
	while (getline(file, line)) 
	{
		lines++;
		if (lines == 0 || lines-1 >= NUM_RECORDS){
			continue;
		}
		column_count = 0;
		stringstream s (line);
		while (getline(s, value, ','))
		{
			if (column_count == 2 && current_location != value){
				location++;
				current_location = value;
				locations[lines - 1] = location;
				cout << "Location #: " << location << "\t| Location: " << current_location << endl;
			}
			else if (column_count == 2 && current_location == value)
			{
				locations[lines - 1] = location;
			}
			else if (column_count == COLUMN_NUMBER_1) 
			{
				input_1[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_2) 
			{
				input_2[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_3) 
			{
				input_3[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_4) 
			{
				input_4[lines - 1] = value.empty() ? 0 : stof(value);
			}
			column_count++;
		}
	}
}

void find_range(float x[], float centroids[], int* max_loc){
	
	//srand(time(0));
	for(int i =0; i < NUM_CLUSTERS; i++){
		int j = rand() % *max_loc; 
		centroids[i] = x[j];
	}
}
__global__ void calculate_centers(float data[], int clusters[], float centers[], int* max_loc){
	__shared__ int counts[NUM_CLUSTERS];
	__shared__ unsigned int temp_counts[NUM_CLUSTERS];
	
	__shared__ float temp_centers[NUM_CLUSTERS];
	
	if(threadIdx.x < NUM_CLUSTERS){
		temp_counts[threadIdx.x] = 0;
		counts[threadIdx.x] = 0;
		temp_centers[threadIdx.x] = 0.0;
		centers[threadIdx.x] = 0.0;
	}
 
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	
	__syncthreads();
	
	while(i < *max_loc){
		atomicAdd(&temp_centers[clusters[i]], data[i]);
		atomicAdd(&temp_counts[clusters[i]], 1);
		i += offset;
	}

	__syncthreads();
	i = threadIdx.x;
	
	if (i < NUM_CLUSTERS){
		atomicAdd(&centers[threadIdx.x], temp_centers[threadIdx.x]);
		atomicAdd(&counts[threadIdx.x], temp_counts[threadIdx.x]);
	}
	
	i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i < NUM_CLUSTERS){
		if(counts[i] != 0){
			centers[i] = centers[i] / counts[i];
		}
	}
}

__global__ void compare(float data[], float centers[], int clusters[], bool* change_cluster, int* max_loc) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;	
	
	float min_diff = abs(data[i]-centers[clusters[i]]);

	while (i < *max_loc ){ 
		for(int j = 0; j < NUM_CLUSTERS; j++){
			float diff = abs(data[i] - centers[j]);
			if (diff < min_diff){
				min_diff = diff;
				
				if( (clusters[i] != j) && (*change_cluster == false) ){
					*change_cluster = true;
				}
				
				clusters[i] = j;
			}
			
			if (abs(min_diff - 0) < 0.0001){
				break;
			}
		}
		i += offset;
	}
}

__global__ void display_data_averages(float data_avg[], int locations[], float data[], int country_count[], float country_avg[], int* max_loc){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while ( i < NUM_RECORDS){
		atomicAdd(&data_avg[locations[i]], data[i]);
		atomicAdd(&country_count[locations[i]], 1);	
		i = i + offset;
	}
	__syncthreads();

	i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < *max_loc){
	    country_avg[i] = data_avg[i] / country_count[i];
	    i = i + offset;
	}
}

__global__ void calculate_correlations(float result_data[], float correlations[]) {
	int xIndex = 0;
	int yIndex = 0;
	int increment = NUM_COLUMNS -1;
	int currIndex = threadIdx.x;

	//calculate the two indices of the data we are comparing using current thread
	while(currIndex >= (NUM_COLUMNS -1)){
	xIndex++;
		increment--;
	if(increment < 1){
		 printf("Error calculating current Indexes to calculate correlations\n");
		 return;
	}
		currIndex -= increment;
	}
	xIndex *= NUM_RECORDS;
	yIndex = (1 + currIndex) * NUM_RECORDS;
	//printf("hello from thread %d. I have xIndex %d and yIndex %d. Current increment is %d. Curr index %d\n", threadIdx.x, xIndex, yIndex, increment, currIndex);

	if(xIndex < 0 || xIndex >= NUM_COLUMNS * NUM_RECORDS || yIndex < 0 || yIndex >= NUM_COLUMNS * NUM_RECORDS || xIndex == yIndex){
	printf("Invalid indices calculated during correlation calculation function\n");
	return;
	}

	__syncthreads();
	if(threadIdx.x < NUM_PAIRS){
		// Calculate mean of each dataset
		float meanx = 0;
		float meany = 0;
		for (int i = 0; i < NUM_RECORDS; i++) {
		meanx = meanx + 0.0001 * result_data[xIndex + i];
		meany = meany + 0.0001 * result_data[yIndex + i];
		}
		meanx = meanx / (NUM_RECORDS * 0.0001);
		meany = meany / (NUM_RECORDS * 0.0001);
		// Calculate deviation scores and product of deviation scores
		float ssx = 0;
		float ssy = 0;
		float xy = 0;
		for (int i = 0; i < NUM_RECORDS; i++) {
		ssx = ssx + 0.0001 * pow(result_data[xIndex + i] - meanx, 2);
		ssy = ssy + 0.0001 * pow(result_data[yIndex + i] - meany, 2);
		xy = xy + 0.0001 * (result_data[xIndex + i] - meanx) * (result_data[yIndex + i] - meany);
		}

		// Calculate correlation
		correlations[threadIdx.x] = (xy / sqrt(ssx * ssy));
		__syncthreads();
	}
	else printf("Invalid thread number\n");
}


__global__ void display_correlations(float correlations[]){
	__syncthreads();
	//float correlation = correlations[threadIdx.x];
	int xIndex = 0;
	int yIndex = 0;
	int increment = NUM_COLUMNS -1;
	int currIndex = threadIdx.x;
	
	while(currIndex >= NUM_COLUMNS -1){
		increment--;
		xIndex++;
		if(increment < 1){
			 printf("Error calculating current Indexes to display correlations\n");
			 return;
		}
		currIndex -= increment;
		}
		xIndex += 1;
		yIndex = (2 + currIndex);

	if (abs(correlations[threadIdx.x]) > 1) {
		printf("Invalid correlation value. Exiting\n");
		return;
	}
	//__syncthreads();
	if(abs(correlations[threadIdx.x]) > 0.7){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a strong positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a strong negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else if(abs(correlations[threadIdx.x]) > 0.5){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a moderate positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a moderate negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else if(abs(correlations[threadIdx.x]) > 0.3){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a weak positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a weak negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else printf("Columns %d and %d have little-to-no correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	return;
}


__global__ void calculate_linear_regressions(float correlations[], float data[])
{
	float sumx = 0;
	float sumy = 0;
	float sumxy = 0;
	float sumxSquared = 0;
	float sumySquared = 0;
	int xIndex = 0;
	int yIndex = 0;
	int increment = NUM_COLUMNS -1;
	int currIndex = threadIdx.x;

	//calculate the two indices of the data we are comparing using current thread
	while(currIndex >= (NUM_COLUMNS -1)){
	xIndex++;
		increment--;
	if(increment < 1){
		 printf("Error calculating current Indexes to calculate linear regressions\n");
		 return;
	}
		currIndex -= increment;
	}
	xIndex *= NUM_RECORDS;
	yIndex = (1 + currIndex) * NUM_RECORDS;
	
	if(abs(correlations[threadIdx.x]) < 0.3){
	printf("Minimal correlation between quanitifiers %d and %d. Skipping Calculating Regression.\n", xIndex / NUM_RECORDS + 1, yIndex / NUM_RECORDS + 1);
	return;
	}

	for (int i = 0; i < NUM_RECORDS; i++) {
		sumx = sumx + 0.0001 * data[xIndex + i];
		sumy = sumy + 0.0001 * data[yIndex + i];
		sumxy = sumxy + 0.0001 * (data[xIndex + i] * data[yIndex + i]);
		sumxSquared = sumxSquared + 0.0001 * pow(data[xIndex + i], 2);
		sumySquared = sumySquared + 0.0001 * pow(data[yIndex + i], 2);
	}
	float a = 10000 * (((NUM_RECORDS * sumxy) - (sumx * sumy)) / ((NUM_RECORDS * sumxSquared) - pow(sumx, 2)));
	float b = 10000 * (((sumy * sumxSquared) - (sumx * sumxy)) / ((NUM_RECORDS * sumxSquared) - pow(sumx, 2)));
	printf("The calculated linear regression for columns %d and %d is %fx + %f\n", xIndex / NUM_RECORDS + 1, yIndex / NUM_RECORDS + 1, a, b);
}

int main() {
	cout << "Starting..." << endl;
	
	 
	float* centers = new float[NUM_CLUSTERS];
	float* input_1 = new float[NUM_RECORDS];
	float* input_2 = new float[NUM_RECORDS];
	float* input_3 = new float[NUM_RECORDS];
	float* input_4 = new float[NUM_RECORDS];
	int* locations = new int[NUM_RECORDS];
	
	
	bool* change_clusters = new bool(true);
	int counter = 20;
	float elapsedTime;

	float *dev_data;
	float *dev_centers;
	int *dev_clusters;
	int* dev_max_loc;
	bool *dev_change_clusters;
	int* dev_locations;
	int* dev_country_count;
	float* dev_data_avg;
	float* dev_country_avg;

	int* index = new int;
	int* mapping = new int[5];

	hipEvent_t	 start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	build_input(input_1, input_2, input_3, input_4, locations);
	
	int* max_loc = max_element(locations,locations+NUM_RECORDS);
	int* clusters = new int[*max_loc];
	float* country_avg = new float[*max_loc];
	int* country_count = new int[*max_loc];
	
	float* data_avg = new float[*max_loc];
	const int SIZE_F = *max_loc * sizeof(float);
	const int SIZE_R = NUM_RECORDS * sizeof(float);

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Read:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	HANDLE_ERROR( hipMalloc( (void**)&dev_data, NUM_RECORDS*sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_data, input_1, NUM_RECORDS*sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_data_avg, SIZE_F) );
	HANDLE_ERROR( hipMemset( dev_data_avg, 0, SIZE_F) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_country_count, *max_loc*sizeof(int) ) );
	HANDLE_ERROR( hipMemset( dev_country_count, 0, *max_loc*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc((void**)&dev_locations, NUM_RECORDS*sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(dev_locations, locations, NUM_RECORDS*sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc((void**)&dev_country_avg, SIZE_F) );
	HANDLE_ERROR( hipMemset(dev_country_avg, 0, SIZE_F) );
	HANDLE_ERROR( hipMalloc((void**)&dev_max_loc, sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(dev_max_loc, max_loc, sizeof(int), hipMemcpyHostToDevice));
	
	display_data_averages<<<1, 1024>>>(dev_data_avg, dev_locations, dev_data, dev_country_count, dev_country_avg, dev_max_loc);
	
	HANDLE_ERROR( hipMemcpy(country_avg, dev_country_avg, *max_loc * sizeof(float), hipMemcpyDeviceToHost) );
	

	find_range(country_avg, centers, max_loc);
	cout << "Original centers:";
	for(int i = 0; i < NUM_CLUSTERS; i++){
		cout << centers[i] << " ";
	}
	cout << endl;
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_centers, NUM_CLUSTERS * sizeof( float ) ) );
	HANDLE_ERROR( hipMemcpy( dev_centers, centers, NUM_CLUSTERS*sizeof(float), hipMemcpyHostToDevice ) ); 
	HANDLE_ERROR( hipMalloc( (void**)&dev_clusters, *max_loc * sizeof( int ) ) );
	HANDLE_ERROR( hipMemset( dev_clusters, 0, *max_loc * sizeof( int )) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_change_clusters, sizeof(bool)));
	HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
							  
	compare<<<1,256>>>(dev_country_avg,dev_centers,dev_clusters,dev_change_clusters,dev_max_loc);

	while( (*change_clusters == true) && (counter > 0) ){
		HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
		
		calculate_centers<<<1,5>>>(dev_country_avg, dev_clusters, dev_centers, dev_max_loc);
		
		HANDLE_ERROR( hipMemcpy(centers, dev_centers, NUM_CLUSTERS*sizeof(float), hipMemcpyDeviceToHost) );
		cout << "Centers: ";
		for(int i = 0; i < NUM_CLUSTERS; i++){
			cout << centers[i] << " ";
		}
		cout << endl;
		
		compare<<<1,20>>>(dev_country_avg,dev_centers,dev_clusters,dev_change_clusters, dev_max_loc);
		
		HANDLE_ERROR( hipMemcpy( change_clusters, dev_change_clusters, sizeof(bool), hipMemcpyDeviceToHost) );
		//cout << "Change_clusters is: " << *change_clusters << endl;
		counter--;
	}
	

	HANDLE_ERROR( hipMemcpy(clusters,dev_clusters, *max_loc*sizeof(int), hipMemcpyDeviceToHost) );
	
	cout << "Final clusters: " << endl;
	for(int i = 0; i < *max_loc; i++){
		cout << clusters[i] << " ";
	}
	cout << endl;

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Analyze:  %3.1f ms\n", elapsedTime );

	int* cluster_count = new int[NUM_CLUSTERS];
	for(int i = 0; i < NUM_CLUSTERS; i++){
		cluster_count[i] = 0;
	}
	for(int i = 0; i < *max_loc; i++){
			cluster_count[clusters[i]]++;
		}

	for(int i = 0; i < NUM_CLUSTERS; i++){
		cout << "Cluster " << i << ": " << cluster_count[i] << endl;
	}
	
	//Results - Ryan

	cout << "Beginning Display of Results\n\n";

	// Allocate device memory
	float *dev_correlations;
	float *dev_result_data;

	float result_data[SIZE_R * NUM_COLUMNS];
	float *correlations = new float[NUM_PAIRS];

	for(int i =0; i < *max_loc; i++){
		if(country_count[i] < 1 || data_avg[i] != data_avg[i]) cout << "No entries seen for country " << i << "\n";
		else cout << "Cluster Average for Country " << i << ":   " << abs(data_avg[i] / country_count[i]) << "\n";
	}
	
	for(int i =0; i < NUM_PAIRS; i++) correlations[i] = 0;
	for(int i = 0; i < NUM_RECORDS; i++){
	    result_data[i] = input_1[i];
    	    result_data[i + NUM_RECORDS] = input_2[i];
	    result_data[i + 2*NUM_RECORDS] = input_3[i];
	    result_data[i+3*NUM_RECORDS] = input_4[i];
	}

	HANDLE_ERROR( hipMalloc( (void**)&dev_result_data, SIZE_R * NUM_COLUMNS ) );
	HANDLE_ERROR( hipMemcpy( dev_result_data, result_data, SIZE_R * NUM_COLUMNS, hipMemcpyHostToDevice) );  

	HANDLE_ERROR( hipMalloc( (void**)&dev_correlations, NUM_PAIRS * sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_correlations, correlations, NUM_PAIRS * sizeof(float), hipMemcpyHostToDevice) );  

	cout << "\n\nCalculating Correlations... ";
	calculate_correlations<<<1, NUM_PAIRS>>>(dev_result_data, dev_correlations);
	hipDeviceSynchronize();

	HANDLE_ERROR( hipMemcpy(correlations, dev_correlations, NUM_PAIRS * sizeof(float), hipMemcpyDeviceToHost));
	cout << "\n\nDisplaying Calculated Correlations: \n\n";

	// Output Correlation Strengths

	display_correlations<<<1, NUM_PAIRS>>>(dev_correlations);
	hipDeviceSynchronize();

	//Calculate Linear Regressions
	cout << "\nCalulating Linear Regressions\n\n";
	calculate_linear_regressions<<<1, NUM_PAIRS>>>(dev_correlations, dev_result_data);
	hipDeviceSynchronize();


	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "\nTime to Output:  %3.1f ms\n", elapsedTime );

	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	hipFree( dev_data );
	hipFree( dev_centers );
	hipFree( dev_clusters );
	hipFree( dev_change_clusters );
	hipFree( dev_max_loc );
	hipFree ( dev_correlations);
	hipFree (dev_data);
	hipFree(dev_country_count);
	
	return 0;
}
