#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdio>
#include <algorithm>
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include "../common/book.h"

using namespace std;

/*
	Column #: Data
	4: total_cases
	5: new_cases
	7: total_deaths
	8: new_deaths
	10: total_cases_per_million
	11: new_cases_per_million
	13: total_deaths_per_million
	14: new_deaths_per_million
	17: icu_patients
	18: icu_patients_per_million
	19: hosp_patients
	20: hosp_patients_per_million
	25: new_tests
	26: total_tests
	31: positive_rate
	32: tests_per_case
	34: total_vaccinations
	35: people_vaccinated
	36: people_fully_vaccinated
	37: total_boosters
	38: new_vaccinations
	48: median_age
*/

const string FILE_NAME = "Data/owid-covid-data-2021-11-03.csv"; // File location
const int COLUMN_NUMBER_1 = 4; // Column #
const int COLUMN_NUMBER_2 = 7; // Column #
const int COLUMN_NUMBER_3 = 31; // Column #
const int COLUMN_NUMBER_4 = 35; // Column #
const int NUM_RECORDS = 130600; // Number of Records to read
const int NUM_CLUSTERS = 5;
const int NUM_PAIRS = 6; //Number of possible Column pairings
const int NUM_COLUMNS = 4; //Number of columns we are using

void build_input(float input_1[], float input_2[], float input_3[], float input_4[], int locations[]){
	ifstream file;
	int lines = -1, location = 0, column_count = 0;
	string line, current_location, value;
	file.open(FILE_NAME);
	while (getline(file, line)) 
	{
		lines++;
		if (lines == 0 || lines-1 >= NUM_RECORDS){
			continue;
		}
		column_count = 0;
		stringstream s (line);
		while (getline(s, value, ','))
		{
			if (column_count == 2 && current_location != value){
				location++;
				current_location = value;
				locations[lines - 1] = location;
			}
			else if (column_count == 2 && current_location == value)
			{
				locations[lines - 1] = location;
			}
			else if (column_count == COLUMN_NUMBER_1) 
			{
				input_1[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_2) 
			{
				input_2[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_3) 
			{
				input_3[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_4) 
			{
				input_4[lines - 1] = value.empty() ? 0 : stof(value);
			}
			column_count++;
		}
	}
}

void find_range(float x[], float centroids[]){
	float* min = min_element(x,x+NUM_RECORDS);
	float* max = max_element(x,x+NUM_RECORDS);
	float range = *max - *min;
	float group_size = range / NUM_CLUSTERS;
	for(int i =0; i < NUM_CLUSTERS; i++){
		centroids[i] = *min + group_size * i;
	}
}

__global__ void calculate_centers(float data[], int clusters[], float centers[]){
	__shared__ int counts[NUM_CLUSTERS];
	__shared__ float x[NUM_RECORDS];
	__shared__ int shared_clusters[NUM_RECORDS];
	__shared__ unsigned int temp_counts[NUM_CLUSTERS];
	
	__shared__ float temp_centers[NUM_CLUSTERS];
	
	if(threadIdx.x < NUM_CLUSTERS){
		temp_counts[threadIdx.x] = 0;
		counts[threadIdx.x] = 0;
		temp_centers[threadIdx.x] = 0.0;
		centers[threadIdx.x] = 0.0;
	}
 
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	
	/*
	if(i < NUM_RECORDS){
		x[i] = data[i];
		shared_clusters[i] = clusters[i];
	}
	*/
	
	/*
	if (i < NUM_CLUSTERS){
		centers[i] = 0.0;
		counts[i] = 0;
	}
	*/
	
	__syncthreads();
	
	while(i < NUM_RECORDS){
		atomicAdd(&temp_centers[clusters[i]], data[i]);
		atomicAdd(&temp_counts[clusters[i]], 1);
		i += offset;
	}

	__syncthreads();
	i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i < NUM_CLUSTERS){
		atomicAdd(&centers[threadIdx.x], temp_centers[threadIdx.x]);
		atomicAdd(&counts[threadIdx.x], temp_counts[threadIdx.x]);
	}
	if (i < NUM_CLUSTERS){
		if(counts[i] != 0){
			centers[i] = centers[i] / counts[i];
		}
	}
}

__global__ void compare(float data[], float centers[], int clusters[], bool* change_cluster) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	float min_diff = abs(data[i]-centers[clusters[i]]);

	if (i < NUM_RECORDS ){
		for(int j = 0; j < NUM_CLUSTERS; j++){
			float diff = abs(data[i] - centers[j]);
			if (diff < min_diff){
				min_diff = diff;
				
				/****NEED TO LOCK?****
				if( (clusters[i] != j) && (*change_cluster == false) ){
					*change_cluster = true;
				}
				*********************/
				
				clusters[i] = j;
			}
			
			if (abs(min_diff - 0) < 0.0001){
				break;
			}
		}
	}
}

__global__ void classify(int clusters[], int locations[], int mapping[], int* max_loc, int* index){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	*index = 0;
	if (i < NUM_RECORDS){
		for(int j = 0; j <= *max_loc; j++){
			if(i < NUM_CLUSTERS){
				mapping[i] = 0;
			}
			
			if(locations[i] == j){
				int x = clusters[i];
				atomicAdd(&mapping[x], 1);
			}
			
			__syncthreads();
			
			if (i == 1){
				int max = mapping[0];
				*index = 0;

				for(int p = 1; p < NUM_CLUSTERS; p++){
					if (mapping[p] > max){
						max = mapping[p];
						*index = p;
					}
				}
			}
			__syncthreads();
			if(locations[i] == j){
				clusters[i] = *index;
			}
		}
	}
}


__global__ void calculate_correlations(float result_data[], float correlations[]) {
    int xIndex = 0;
    int yIndex = 0;
    int increment = NUM_COLUMNS -1;
    int currIndex = threadIdx.x;

    //calculate the two indices of the data we are comparing using current thread
    while(currIndex >= (NUM_COLUMNS -1)){
	xIndex++;
        increment--;
	if(increment < 1){
	     printf("Error calculating current Indexes to calculate correlations\n");
	     return;
	}
        currIndex -= increment;
    }
    xIndex *= NUM_RECORDS;
    yIndex = (1 + currIndex) * NUM_RECORDS;
    //printf("hello from thread %d. I have xIndex %d and yIndex %d. Current increment is %d. Curr index %d\n", threadIdx.x, xIndex, yIndex, increment, currIndex);

    if(xIndex < 0 || xIndex >= NUM_COLUMNS * NUM_RECORDS || yIndex < 0 || yIndex >= NUM_COLUMNS * NUM_RECORDS || xIndex == yIndex){
	printf("Invalid indices calculated during correlation calculation function\n");
	return;
    }

    __syncthreads();
    if(threadIdx.x < NUM_PAIRS){
	    // Calculate mean of each dataset
	    float meanx = 0;
	    float meany = 0;
	    for (int i = 0; i < NUM_RECORDS; i++) {
		meanx = meanx + 0.0001 * result_data[xIndex + i];
		meany = meany + 0.0001 * result_data[yIndex + i];
	    }
	    meanx = meanx / (NUM_RECORDS * 0.0001);
	    meany = meany / (NUM_RECORDS * 0.0001);
	    // Calculate deviation scores and product of deviation scores
	    float ssx = 0;
	    float ssy = 0;
	    float xy = 0;
	    for (int i = 0; i < NUM_RECORDS; i++) {
		ssx = ssx + 0.0001 * pow(result_data[xIndex + i] - meanx, 2);
		ssy = ssy + 0.0001 * pow(result_data[yIndex + i] - meany, 2);
		xy = xy + 0.0001 * (result_data[xIndex + i] - meanx) * (result_data[yIndex + i] - meany);
	    }

	    // Calculate correlation
	    correlations[threadIdx.x] = (xy / sqrt(ssx * ssy));
	    __syncthreads();
    }
    else printf("Invalid thread number\n");
}


__global__ void display_correlations(float correlations[]){
	__syncthreads();
	//float correlation = correlations[threadIdx.x];
	int xIndex = 0;
	int yIndex = 0;
	int increment = NUM_COLUMNS -1;
	int currIndex = threadIdx.x;
	
	while(currIndex >= NUM_COLUMNS -1){
	    increment--;
	    xIndex++;
	    if(increment < 1){
	         printf("Error calculating current Indexes to display correlations\n");
	         return;
	    }
        currIndex -= increment;
        }
        xIndex += 1;
        yIndex = (2 + currIndex);

	if (abs(correlations[threadIdx.x]) > 1) {
		printf("Invalid correlation value. Exiting\n");
		return;
	}
	//__syncthreads();
	if(abs(correlations[threadIdx.x]) > 0.7){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a strong positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a strong negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else if(abs(correlations[threadIdx.x]) > 0.5){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a moderate positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a moderate negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else if(abs(correlations[threadIdx.x]) > 0.3){
		if(correlations[threadIdx.x] > 0) printf("Columns %d and %d have a weak positive correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
		else printf("Columns %d and %d have a weak negative correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	}
	else printf("Columns %d and %d have little-to-no correlation of %f\n", xIndex, yIndex, correlations[threadIdx.x]);
	return;
}


__global__ void calculate_linear_regressions(float correlations[], float data[])
{
    float sumx = 0;
    float sumy = 0;
    float sumxy = 0;
    float sumxSquared = 0;
    float sumySquared = 0;
    int xIndex = 0;
    int yIndex = 0;
    int increment = NUM_COLUMNS -1;
    int currIndex = threadIdx.x;

    //calculate the two indices of the data we are comparing using current thread
    while(currIndex >= (NUM_COLUMNS -1)){
	xIndex++;
        increment--;
	if(increment < 1){
	     printf("Error calculating current Indexes to calculate linear regressions\n");
	     return;
	}
        currIndex -= increment;
    }
    xIndex *= NUM_RECORDS;
    yIndex = (1 + currIndex) * NUM_RECORDS;
    
    if(abs(correlations[threadIdx.x]) < 0.3){
	printf("Minimal correlation between quanitifiers %d and %d. Skipping Calculating Regression.\n", xIndex / NUM_RECORDS + 1, yIndex / NUM_RECORDS + 1);
	return;
    }

    for (int i = 0; i < NUM_RECORDS; i++) {
        sumx = sumx + 0.0001 * data[xIndex + i];
        sumy = sumy + 0.0001 * data[yIndex + i];
        sumxy = sumxy + 0.0001 * (data[xIndex + i] * data[yIndex + i]);
        sumxSquared = sumxSquared + 0.0001 * pow(data[xIndex + i], 2);
        sumySquared = sumySquared + 0.0001 * pow(data[yIndex + i], 2);
    }
    float a = 10000 * (((NUM_RECORDS * sumxy) - (sumx * sumy)) / ((NUM_RECORDS * sumxSquared) - pow(sumx, 2)));
    float b = 10000 * (((sumy * sumxSquared) - (sumx * sumxy)) / ((NUM_RECORDS * sumxSquared) - pow(sumx, 2)));
    printf("The calculated linear regression for columns %d and %d is %fx + %f\n", xIndex / NUM_RECORDS + 1, yIndex / NUM_RECORDS + 1, a, b);
}


__global__ void display_cluster_averages(float cluster_avg[], int locations[], int clusters[], int country_count[]){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if( i < NUM_RECORDS){
	    atomicAdd(&cluster_avg[locations[i]], clusters[i]);
	    atomicAdd(&country_count[i], 1);	
	}
	__syncthreads();
}

int main() {
	cout << "Starting..." << endl;
	
	const int SIZE_F = NUM_RECORDS * sizeof(float); 
	float* centers = new float[NUM_CLUSTERS];
	float* input_1 = new float[NUM_RECORDS];
	float* input_2 = new float[NUM_RECORDS];
	float* input_3 = new float[NUM_RECORDS];
	float* input_4 = new float[NUM_RECORDS];
	int* locations = new int[NUM_RECORDS];
	int* clusters = new int[NUM_RECORDS];
	bool* change_clusters = new bool(true);
	int counter = 20;
	float elapsedTime;

	float *dev_data;
	float *dev_centers;
	int *dev_clusters;
	int* dev_max_loc;
	bool *dev_change_clusters;
	int* dev_mapping;
	int* dev_locations;
	int* dev_index;

	int* index = new int;
	int* mapping = new int[5];

	hipEvent_t	 start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	build_input(input_1, input_2, input_3, input_4, locations);

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Read:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );
	
	int* max_loc = max_element(locations,locations+NUM_RECORDS);

	find_range(input_1, centers);
	//cout << "Original centers:";
	//for(int i = 0; i < NUM_CLUSTERS; i++){
	//	cout << centers[i] << " ";
	//}
	//cout << endl;

	HANDLE_ERROR( hipMalloc( (void**)&dev_data, SIZE_F ) );
	HANDLE_ERROR( hipMemcpy( dev_data, input_1, SIZE_F, hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_centers, NUM_CLUSTERS * sizeof( float ) ) );
	HANDLE_ERROR( hipMemcpy( dev_centers, centers, NUM_CLUSTERS*sizeof(float), hipMemcpyHostToDevice ) ); 
	HANDLE_ERROR( hipMalloc( (void**)&dev_clusters, NUM_RECORDS * sizeof( int ) ) );
	HANDLE_ERROR( hipMemset( dev_clusters, 0, NUM_RECORDS * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_change_clusters, sizeof(bool)));
	HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
							  
	compare<<<1,20>>>(dev_data,dev_centers,dev_clusters,dev_change_clusters);
	
	HANDLE_ERROR( hipMemcpy(clusters, dev_clusters, NUM_RECORDS*sizeof(int), hipMemcpyDeviceToHost));
	//cout << "Clusters: ";
	//for(int i = 0; i < NUM_RECORDS; i++){
	//	cout << clusters[i] << " ";
	//}
	//cout << endl;
	
	while( (*change_clusters == true) && (counter > 0) ){
		HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
		
		calculate_centers<<<1,5>>>(dev_data, dev_clusters, dev_centers);
		
		HANDLE_ERROR( hipMemcpy(centers, dev_centers, NUM_CLUSTERS*sizeof(float), hipMemcpyDeviceToHost) );
		//cout << "Centers: ";
		//for(int i = 0; i < NUM_CLUSTERS; i++){
		//	cout << centers[i] << " ";
		//}
		//cout << endl;
		compare<<<1,20>>>(dev_data,dev_centers,dev_clusters,dev_change_clusters);
		
		HANDLE_ERROR( hipMemcpy( clusters, dev_clusters, NUM_RECORDS*sizeof(float), hipMemcpyDeviceToHost) );
		//cout << "Clusters: ";
		//for(int i = 0; i < NUM_RECORDS; i++){
		//	cout << clusters[i] << " ";
		//}
		//cout << endl;
		HANDLE_ERROR( hipMemcpy( change_clusters, dev_change_clusters, sizeof(bool), hipMemcpyDeviceToHost) );
		//cout << "Change_clusters is: " << *change_clusters << endl;
		//counter--;
	}
	
	HANDLE_ERROR( hipMalloc((void**)&dev_locations, NUM_RECORDS*sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(dev_locations, locations, NUM_RECORDS*sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc((void**)&dev_mapping, NUM_CLUSTERS*sizeof(int)));
	HANDLE_ERROR( hipMemset(dev_mapping, 0, sizeof(int)) );
	HANDLE_ERROR( hipMalloc((void**)&dev_max_loc,sizeof(int)));
	HANDLE_ERROR( hipMemcpy(dev_max_loc, max_loc, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMalloc((void**)&dev_index, sizeof(int)) );
	HANDLE_ERROR( hipMemset(dev_index, 0, sizeof(int)) );
	
	classify<<<1,20>>>(dev_clusters,dev_locations,dev_mapping,dev_max_loc,dev_index);

	HANDLE_ERROR( hipMemcpy(clusters,dev_clusters, NUM_RECORDS*sizeof(int), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(mapping,dev_mapping, NUM_CLUSTERS*sizeof(int), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(index, dev_index, sizeof(int), hipMemcpyDeviceToHost) );
	
	cout << "Final clusters: " << endl;
	//for(int i = 0; i < 100; i++){
	//	cout << clusters[i] << ":-> ";
	//}
	//cout << endl;

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Analyze:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );
	
	//Results - Ryan

	cout << "Beginning Display of Results\n\n";

	// Allocate device memory
        float *dev_correlations;
        float *dev_result_data;
	float *dev_cluster_avg;
	int *dev_country_count;

	float result_data[SIZE_F * NUM_COLUMNS];
	float *correlations = new float[NUM_PAIRS];
	float* cluster_avg = new float[*max_loc];
	int* country_count = new int[*max_loc];

	HANDLE_ERROR( hipMalloc( (void**)&dev_cluster_avg, *max_loc * sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_cluster_avg, cluster_avg, *max_loc * sizeof(float) , hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMalloc( (void**)&dev_country_count, *max_loc * sizeof(int) ) );
	HANDLE_ERROR( hipMemcpy( dev_country_count, country_count, *max_loc * sizeof(int) , hipMemcpyHostToDevice) );

	display_cluster_averages<<<1, 20>>>(dev_cluster_avg, dev_locations, dev_clusters, dev_country_count);
	hipDeviceSynchronize();

	HANDLE_ERROR( hipMemcpy(cluster_avg, dev_cluster_avg, NUM_PAIRS * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(country_count, dev_country_count, NUM_PAIRS * sizeof(float), hipMemcpyDeviceToHost));
	for(int i =0; i < *max_loc; i++){
	    if(country_count[i] < 1) cout << "No entries seen for country " << i << "\n";
	    else cout << "Cluster Average for Country " << i << ":   " << cluster_avg[i] / country_count[i] << "\n";
	}
        for(int i =0; i < NUM_PAIRS; i++) correlations[i] = 0;
	for(int i = 0; i < NUM_RECORDS; i++) result_data[i] = input_1[i];
	for(int i = 0; i < NUM_RECORDS; i++) result_data[i + NUM_RECORDS] = input_2[i];
	for(int i = 0; i < NUM_RECORDS; i++) result_data[i + 2*NUM_RECORDS] = input_3[i];
	for(int i = 0; i < NUM_RECORDS; i++) result_data[i+3*NUM_RECORDS] = input_4[i];

	HANDLE_ERROR( hipMalloc( (void**)&dev_result_data, SIZE_F * NUM_COLUMNS ) );
	HANDLE_ERROR( hipMemcpy( dev_result_data, result_data, SIZE_F * NUM_COLUMNS, hipMemcpyHostToDevice) );  

	HANDLE_ERROR( hipMalloc( (void**)&dev_correlations, NUM_PAIRS * sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_correlations, correlations, NUM_PAIRS * sizeof(float), hipMemcpyHostToDevice) );  

	cout << "Calculating Correlations... ";
	calculate_correlations<<<1, NUM_PAIRS>>>(dev_result_data, dev_correlations);
	hipDeviceSynchronize();

	HANDLE_ERROR( hipMemcpy(correlations, dev_correlations, NUM_PAIRS * sizeof(float), hipMemcpyDeviceToHost));
	cout << "\n\nDisplaying Calculated Correlations: \n\n";

	// Output Correlation Strengths

	display_correlations<<<1, NUM_PAIRS>>>(dev_correlations);
	hipDeviceSynchronize();

	//Calculate Linear Regressions
	cout << "\nCalulating Linear Regressions\n\n";
	calculate_linear_regressions<<<1, NUM_PAIRS>>>(dev_correlations, dev_result_data);
	hipDeviceSynchronize();


	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "\nTime to Output:  %3.1f ms\n", elapsedTime );

	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	hipFree( dev_data );
	hipFree( dev_centers );
	hipFree( dev_clusters );
	hipFree( dev_change_clusters );
	hipFree( dev_max_loc );
	hipFree ( dev_correlations);
	hipFree (dev_data);
	hipFree(dev_country_count);
	hipFree(dev_cluster_avg);
	
	return 0;
}
