#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdio>
#include <algorithm>
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include "../common/book.h"

using namespace std;

/*
	Column #: Data
	4: total_cases
	5: new_cases
	7: total_deaths
	8: new_deaths
	10: total_cases_per_million
	11: new_cases_per_million
	13: total_deaths_per_million
	14: new_deaths_per_million
	17: icu_patients
	18: icu_patients_per_million
	19: hosp_patients
	20: hosp_patients_per_million
	25: new_tests
	26: total_tests
	31: positive_rate
	32: tests_per_case
	34: total_vaccinations
	35: people_vaccinated
	36: people_fully_vaccinated
	37: total_boosters
	38: new_vaccinations
	48: median_age
*/

const string FILE_NAME = "Data/owid-covid-data-2021-11-03.csv"; // File location
const int COLUMN_NUMBER_1 = 4; // Column #
const int COLUMN_NUMBER_2 = 7; // Column #
const int COLUMN_NUMBER_3 = 31; // Column #
const int COLUMN_NUMBER_4 = 35; // Column #
const int NUM_RECORDS = 130600; // Number of Records to read
const int NUM_CLUSTERS = 5;

void build_input(float input_1[], float input_2[], float input_3[], float input_4[], int locations[]){
	ifstream file;
	int lines = -1, location = 0, column_count = 0;
	string line, current_location, value;
	file.open(FILE_NAME);
	while (getline(file, line)) 
	{
		lines++;
		if (lines == 0 || lines-1 >= NUM_RECORDS){
			continue;
		}
		column_count = 0;
		stringstream s (line);
		while (getline(s, value, ','))
		{
			if (column_count == 2 && current_location != value){
				location++;
				current_location = value;
				locations[lines - 1] = location;
			}
			else if (column_count == 2 && current_location == value)
			{
				locations[lines - 1] = location;
			}
			else if (column_count == COLUMN_NUMBER_1) 
			{
				input_1[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_2) 
			{
				input_2[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_3) 
			{
				input_3[lines - 1] = value.empty() ? 0 : stof(value);
			}
			else if (column_count == COLUMN_NUMBER_4) 
			{
				input_4[lines - 1] = value.empty() ? 0 : stof(value);
			}
			column_count++;
		}
	}
}

void find_range(float x[], float centroids[]){
	float* min = min_element(x,x+NUM_RECORDS);
	float* max = max_element(x,x+NUM_RECORDS);
	float range = *max - *min;
	float group_size = range / NUM_CLUSTERS;
	for(int i =0; i < NUM_CLUSTERS; i++){
		centroids[i] = *min + group_size * i;
	}
}

__global__ void calculate_centers(float data[], int clusters[], float centers[]){
	__shared__ int counts[NUM_CLUSTERS];
	__shared__ float x[NUM_RECORDS];
	__shared__ int shared_clusters[NUM_RECORDS];
	__shared__ unsigned int temp_counts[NUM_CLUSTERS];
	
	__shared__ float temp_centers[NUM_CLUSTERS];
	
	if(threadIdx.x < NUM_CLUSTERS){
		temp_counts[threadIdx.x] = 0;
		counts[threadIdx.x] = 0;
		temp_centers[threadIdx.x] = 0.0;
		centers[threadIdx.x] = 0.0;
	}
 
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	
	/*
	if(i < NUM_RECORDS){
		x[i] = data[i];
		shared_clusters[i] = clusters[i];
	}
	*/
	
	/*
	if (i < NUM_CLUSTERS){
		centers[i] = 0.0;
		counts[i] = 0;
	}
	*/
	
	__syncthreads();
	
	while(i < NUM_RECORDS){
		atomicAdd(&temp_centers[clusters[i]], data[i]);
		atomicAdd(&temp_counts[clusters[i]], 1);
		i += offset;
	}

	__syncthreads();
	i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i < NUM_CLUSTERS){
		atomicAdd(&centers[threadIdx.x], temp_centers[threadIdx.x]);
		atomicAdd(&counts[threadIdx.x], temp_counts[threadIdx.x]);
	}
	if (i < NUM_CLUSTERS){
		if(counts[i] != 0){
			centers[i] = centers[i] / counts[i];
		}
	}
}

__global__ void compare(float data[], float centers[], int clusters[], bool* change_cluster) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	float min_diff = abs(data[i]-centers[clusters[i]]);

	if (i < NUM_RECORDS ){
		for(int j = 0; j < NUM_CLUSTERS; j++){
			float diff = abs(data[i] - centers[j]);
			if (diff < min_diff){
				min_diff = diff;
				
				/****NEED TO LOCK?****
				if( (clusters[i] != j) && (*change_cluster == false) ){
					*change_cluster = true;
				}
				*********************/
				
				clusters[i] = j;
			}
			
			if (abs(min_diff - 0) < 0.0001){
				break;
			}
		}
	}
}

__global__ void classify(int clusters[], int locations[], int mapping[], int* max_loc, int* index){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	*index = 0;
	if (i < NUM_RECORDS){
		for(int j = 0; j <= *max_loc; j++){
			if(i < NUM_CLUSTERS){
				mapping[i] = 0;
			}
			
			if(locations[i] == j){
				int x = clusters[i];
				atomicAdd(&mapping[x], 1);
			}
			
			__syncthreads();
			
			if (i == 1){
				int max = mapping[0];
				*index = 0;

				for(int p = 1; p < NUM_CLUSTERS; p++){
					if (mapping[p] > max){
						max = mapping[p];
						*index = p;
					}
				}
			}
			__syncthreads();
			if(locations[i] == j){
				clusters[i] = *index;
			}
		}
	}
}	

int main() {
	cout << "Starting..." << endl;
	
	const int SIZE_F = NUM_RECORDS * sizeof(float); 
	float* centers = new float[NUM_CLUSTERS];
	float* input_1 = new float[NUM_RECORDS];
	float* input_2 = new float[NUM_RECORDS];
	float* input_3 = new float[NUM_RECORDS];
	float* input_4 = new float[NUM_RECORDS];
	int* locations = new int[NUM_RECORDS];
	int* clusters = new int[NUM_RECORDS];
	bool* change_clusters = new bool(true);
	int counter = 20;
	float elapsedTime;

	float *dev_data;
	float *dev_centers;
	int *dev_clusters;
	int* dev_max_loc;
	bool *dev_change_clusters;
	int* dev_mapping;
	int* dev_locations;
	int* dev_index;

	int* index = new int;
	int* mapping = new int[5];

	hipEvent_t	 start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	build_input(input_1, input_2, input_3, input_4, locations);

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Read:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );
	
	int* max_loc = max_element(locations,locations+NUM_RECORDS);

	find_range(input_1, centers);
	//cout << "Original centers:";
	//for(int i = 0; i < NUM_CLUSTERS; i++){
	//	cout << centers[i] << " ";
	//}
	//cout << endl;

	HANDLE_ERROR( hipMalloc( (void**)&dev_data, SIZE_F ) );
	HANDLE_ERROR( hipMemcpy( dev_data, input_1, SIZE_F, hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_centers, NUM_CLUSTERS * sizeof( float ) ) );
	HANDLE_ERROR( hipMemcpy( dev_centers, centers, NUM_CLUSTERS*sizeof(float), hipMemcpyHostToDevice ) ); 
	HANDLE_ERROR( hipMalloc( (void**)&dev_clusters, NUM_RECORDS * sizeof( int ) ) );
	HANDLE_ERROR( hipMemset( dev_clusters, 0, NUM_RECORDS * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_change_clusters, sizeof(bool)));
	HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
							  
	compare<<<1,20>>>(dev_data,dev_centers,dev_clusters,dev_change_clusters);
	
	HANDLE_ERROR( hipMemcpy(clusters, dev_clusters, NUM_RECORDS*sizeof(int), hipMemcpyDeviceToHost));
	//cout << "Clusters: ";
	//for(int i = 0; i < NUM_RECORDS; i++){
	//	cout << clusters[i] << " ";
	//}
	//cout << endl;
	
	while( (*change_clusters == true) && (counter > 0) ){
		HANDLE_ERROR( hipMemset( dev_change_clusters, false, sizeof(bool)) );
		
		calculate_centers<<<1,5>>>(dev_data, dev_clusters, dev_centers);
		
		HANDLE_ERROR( hipMemcpy(centers, dev_centers, NUM_CLUSTERS*sizeof(float), hipMemcpyDeviceToHost) );
		//cout << "Centers: ";
		//for(int i = 0; i < NUM_CLUSTERS; i++){
		//	cout << centers[i] << " ";
		//}
		//cout << endl;
		compare<<<1,20>>>(dev_data,dev_centers,dev_clusters,dev_change_clusters);
		
		HANDLE_ERROR( hipMemcpy( clusters, dev_clusters, NUM_RECORDS*sizeof(float), hipMemcpyDeviceToHost) );
		//cout << "Clusters: ";
		//for(int i = 0; i < NUM_RECORDS; i++){
		//	cout << clusters[i] << " ";
		//}
		//cout << endl;
		HANDLE_ERROR( hipMemcpy( change_clusters, dev_change_clusters, sizeof(bool), hipMemcpyDeviceToHost) );
		//cout << "Change_clusters is: " << *change_clusters << endl;
		//counter--;
	}
	
	HANDLE_ERROR( hipMalloc((void**)&dev_locations, NUM_RECORDS*sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(dev_locations, locations, NUM_RECORDS*sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc((void**)&dev_mapping, NUM_CLUSTERS*sizeof(int)));
	HANDLE_ERROR( hipMemset(dev_mapping, 0, sizeof(int)) );
	HANDLE_ERROR( hipMalloc((void**)&dev_max_loc,sizeof(int)));
	HANDLE_ERROR( hipMemcpy(dev_max_loc, max_loc, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMalloc((void**)&dev_index, sizeof(int)) );
	HANDLE_ERROR( hipMemset(dev_index, 0, sizeof(int)) );
	
	classify<<<1,20>>>(dev_clusters,dev_locations,dev_mapping,dev_max_loc,dev_index);

	HANDLE_ERROR( hipMemcpy(clusters,dev_clusters, NUM_RECORDS*sizeof(int), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(mapping,dev_mapping, NUM_CLUSTERS*sizeof(int), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(index, dev_index, sizeof(int), hipMemcpyDeviceToHost) );
	
	//cout << "Final clusters: " << endl;
	//for(int i = 0; i < NUM_RECORDS; i++){
	//	cout << clusters[i] << " ";
	//}
	//cout << endl;

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Analyze:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	//Ryan

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to Output:  %3.1f ms\n", elapsedTime );

	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	hipFree( dev_data );
	hipFree( dev_centers );
	hipFree( dev_clusters );
	hipFree( dev_change_clusters );
	hipFree( dev_max_loc );
	
	return 0;
}
